#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <complex>
#include "../config/config.hpp"
#include "mimo_synthesis.cuh"

namespace MIMOSynthesis {

    __global__ void synthesize_peaks_kernel(const hipDoubleComplex* d_data, RadarData::Peak* d_peakList,hipDoubleComplex* d_peaksnaps, int num_peaks, int num_receivers, int num_chirps, int num_samples, int max_num_peaks) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_peaks) return;
        // Extract peak information
        RadarData::Peak peak = d_peakList[idx];
        // Validate indices
        if (peak.receiver < 0 || peak.receiver >= num_receivers ||
            peak.chirp < 0 || peak.chirp >= num_chirps ||
            peak.sample < 0 || peak.sample >= num_samples) {
            return; // Invalid peak indices
        }
        
        for (int r = 0; r < num_receivers; ++r) {
            int i = r * num_chirps * num_samples + peak.chirp * num_samples + peak.sample;
            d_peaksnaps[idx * num_receivers + r] = d_data[i];
        }
    }
    void synthesize_peaks(const RadarData::Frame& frame, RadarData::peakInfo& peakinfo) {
        // Clear the output PeakSnaps
        peakinfo.initializePeakSnaps();
        int blocks = (peakinfo.num_peaks + 255) / 256; // Calculate number of blocks
        int threads = 256; // Number of threads per block
        std::cout << "Number of blocks: " << blocks << ", Threads per block: " << threads << std::endl;
        synthesize_peaks_kernel<<<blocks,threads>>>(frame.d_data, peakinfo.d_peakList,peakinfo.d_peaksnaps, peakinfo.num_peaks, frame.num_receivers, frame.num_chirps, frame.num_samples, peakinfo.max_num_peaks);
        hipDeviceSynchronize();
        // Check for CUDA errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA error in synthesize_peaks_kernel: " << hipGetErrorString(err) << std::endl;
            return;
        }
    }
}
