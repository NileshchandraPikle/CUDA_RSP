#include "datatypes.cuh"
#include <fstream>
#include <sstream>
#include <iostream>
#include <cstring>
#include "../cuda_utils/cuda_utils.hpp"

namespace RadarData {
EgoEstimationOutput::EgoEstimationOutput() : d_sum(nullptr), d_count(nullptr) {}
EgoEstimationOutput::~EgoEstimationOutput() { free(); }
void EgoEstimationOutput::allocate() {
    hipMalloc(&d_sum, sizeof(double));
    hipMalloc(&d_count, sizeof(int));
}
void EgoEstimationOutput::free() {
    if (d_sum) hipFree(d_sum);
    if (d_count) hipFree(d_count);
    d_sum = nullptr;
    d_count = nullptr;
}
void EgoEstimationOutput::zero(hipStream_t stream) {
    hipMemsetAsync(d_sum, 0, sizeof(double), stream);
    hipMemsetAsync(d_count, 0, sizeof(int), stream);
}
void EgoEstimationOutput::copy_to_host(double& h_sum, int& h_count, hipStream_t stream) const {
    hipMemcpyAsync(&h_sum, d_sum, sizeof(double), hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
}

Frame::Frame(int r, int c, int s)
    : num_receivers(r), num_chirps(c), num_samples(s), d_data(nullptr)
{
    data = new Complex[r * c * s]();
    allocate_frame_mem_device();
}

Frame::~Frame() {
    delete[] data;
    free_device();
}

Complex& Frame::operator()(int receiver, int chirp, int sample) {
    return data[idx(receiver, chirp, sample)];
}
const Complex& Frame::operator()(int receiver, int chirp, int sample) const {
    return data[idx(receiver, chirp, sample)];
}

// Device memory management
void Frame::allocate_frame_mem_device() {
    if (!d_data) {
        size_t total = num_receivers * num_chirps * num_samples;
        CUDA_CHECK(hipMalloc(&d_data, total * sizeof(hipDoubleComplex)));
    }
}
void Frame::free_device() {
    if (d_data) {
        CUDA_CHECK(hipFree(d_data));
        d_data = nullptr;
    }
}
void Frame::copy_frame_to_device() {
    size_t total = num_receivers * num_chirps * num_samples;
    CUDA_CHECK(hipMemcpy(
    d_data,
    reinterpret_cast<const hipDoubleComplex*>(data),
    total * sizeof(hipDoubleComplex),
    hipMemcpyHostToDevice));
    //std::cout << "Frame Data copied to device" << std::endl;
}
void Frame::copy_frame_to_host() {
    if (d_data) {
        size_t total = num_receivers * num_chirps * num_samples;
       CUDA_CHECK(hipMemcpy(
    reinterpret_cast<hipDoubleComplex*>(data),
    d_data,
    total * sizeof(hipDoubleComplex),
    hipMemcpyDeviceToHost));
    }
}

// Initialize frame with data from CSV
void initialize_frame(Frame& frame, int num_receivers, int num_chirps, int num_samples, int frameIndex) {
    //Frame frame(num_receivers, num_chirps, num_samples);

    std::ifstream file("/mnt/mydisk/Nilesh/CUDA_RSP/data/radar_indexed.csv");
    if (!file.is_open()) {
        std::cerr << "Error: Could not open radar_indexed.csv" << std::endl;
        return;
    }

    std::string line;
    bool frameDataLoaded = false;
    while (std::getline(file, line)) {
        std::istringstream ss(line);
        int frame_number, receiver, chirp, sample;
        double value;
        char delimiter;
        ss >> frame_number >> delimiter >> receiver >> delimiter >> chirp >> delimiter >> sample >> delimiter >> value;

        if (frame_number == frameIndex) {
            if (receiver < num_receivers && chirp < num_chirps && sample < num_samples) {
                frame(receiver, chirp, sample) = Complex(value, 0);
            }
            frameDataLoaded = true;
        } else if (frameDataLoaded) {
            break;
        }
    }
    file.close();
    //return frame;
}

size_t frame_size_bytes(const Frame& frame) {
    return static_cast<size_t>(frame.num_receivers) *
           frame.num_chirps *
           frame.num_samples *
           sizeof(Complex);
}

// Initialize multiple frames for batch processing
size_t initializeBatchFrames(
    std::vector<Frame>& frames, 
    int numFrames,
    int num_receivers,
    int num_chirps, 
    int num_samples
) {
    // Clear the vector in case it has existing frames
    frames.clear();
    frames.reserve(numFrames);
    
    std::cout << "Initializing " << numFrames << " frames for batch processing..." << std::endl;
    
    // Calculate memory requirements
    size_t mem_per_frame = sizeof(std::complex<double>) * num_receivers * num_chirps * num_samples;
    size_t total_mem = mem_per_frame * numFrames;
    std::cout << "Memory per frame: " << mem_per_frame / (1024*1024) << " MB" << std::endl;
    std::cout << "Total memory for batch: " << total_mem / (1024*1024) << " MB" << std::endl;
    
    // Check if we have enough GPU memory
    size_t free_mem = 0, total_mem_gpu = 0;
    hipMemGetInfo(&free_mem, &total_mem_gpu);
    std::cout << "GPU memory: " << total_mem_gpu / (1024*1024) << " MB total, " 
              << free_mem / (1024*1024) << " MB free" << std::endl;
    
    try {
        for (int i = 0; i < numFrames; ++i) {
            std::cout << "Initializing frame " << i+1 << "/" << numFrames << "..." << std::endl;
            
            // Create a new frame
            frames.emplace_back(num_receivers, num_chirps, num_samples);
            
            // Initialize frame with data
            initialize_frame(
                frames.back(),
                num_receivers,
                num_chirps,
                num_samples,
                i % numFrames // Cycle through available frame indices
            );
            
            std::cout << "Copying frame " << i+1 << " to device..." << std::endl;
            // Copy to device
            frames.back().copy_frame_to_device();
            std::cout << "Frame " << i+1 << " initialized and copied to device." << std::endl;
        }
    } catch (const std::exception& e) {
        std::cerr << "Exception during frame initialization: " << e.what() << std::endl;
        throw;
    }
    
    // Calculate and return frame size in bytes for a single frame
    size_t frame_size = frame_size_bytes(frames[0]);
    std::cout << "Single frame size: " << frame_size << " bytes" << std::endl;
    std::cout << "Total batch size: " << (frame_size * numFrames) / (1024*1024) << " MB" << std::endl;
    
    return frame_size * numFrames;
}

peakInfo::peakInfo(int r, int c, int s)
{
    num_receivers = r;
    num_chirps = c;
    num_samples = s;
    std::cout << "Creating peakInfo with dimensions: "
              << num_receivers << " receivers, "
              << num_chirps << " chirps, "
              << num_samples << " samples." << std::endl;
    value = 0.0;
    num_peaks = 0; // Initialize number of peaks to zero
     // Device variable to hold number of peaks
    max_num_peaks = num_receivers*num_chirps*num_samples; // Default value, can be adjusted as needed
    
    nci = nullptr;
    foldedNci = nullptr;
    noiseEstimation = nullptr;
    thresholdingMap = nullptr;
    peakList = nullptr;
    peaksnaps = nullptr;    
    
    d_nci = nullptr;
    d_foldedNci = nullptr;
    d_noiseEstimation = nullptr;
    d_thresholdingMap = nullptr;
    d_peakList = nullptr;
    d_num_peaks = nullptr;
    d_peak_counter = nullptr;
    d_peaksnaps = nullptr;
    allocate_peakInfo_mem_host();
    allocate_peakInfo_mem_device();
}
peakInfo::~peakInfo() {
    free_peakInfo_host();
    free_peakInfo_device();
}
void peakInfo::allocate_peakInfo_mem_host() {
    int size = num_chirps * num_samples;
    //std::cout << "Allocating memory for peakInfo on host: " << size << " elements." << std::endl;
    if (!nci) {
        nci = new double[size];
        memset(nci, 0, size * sizeof(double));
    }
    if (!foldedNci) {
        foldedNci = new double[size];
        memset(foldedNci, 0, size * sizeof(double));
    }
    if (!noiseEstimation) {
        noiseEstimation = new double[size];
        memset(noiseEstimation, 0, size * sizeof(double));
    }
    if (!thresholdingMap) {
        thresholdingMap = new double[size];
        memset(thresholdingMap, 0, size * sizeof(double));
    }
    if (!peakList) {
        peakList = new Peak[max_num_peaks];
        memset(peakList, 0, max_num_peaks * sizeof(Peak));
    }
} //allocate_peakInfo_mem_host
void peakInfo::free_peakInfo_host() {
    delete[] nci;
    delete[] foldedNci;
    delete[] noiseEstimation;
    delete[] thresholdingMap;
    delete[] peakList;

    nci = nullptr;
    foldedNci = nullptr;
    noiseEstimation = nullptr;
    thresholdingMap = nullptr;
    peakList = nullptr;
}// free_peakInfo_host

void peakInfo::allocate_peakInfo_mem_device() {
    int size = num_chirps * num_samples;
    if(!d_peak_counter){
        CUDA_CHECK(hipMalloc(&d_peak_counter, sizeof(int)));
        CUDA_CHECK(hipMemset(d_peak_counter, 0, sizeof(int)));
    }
    if (!d_nci) {
        CUDA_CHECK(hipMalloc(&d_nci, size * sizeof(double)));
        CUDA_CHECK(hipMemset(d_nci, 0, size * sizeof(double)));
    }
    if (!d_foldedNci) {
        CUDA_CHECK(hipMalloc(&d_foldedNci, size * sizeof(double)));
        CUDA_CHECK(hipMemset(d_foldedNci, 0, size * sizeof(double)));
    }
    if (!d_noiseEstimation) {
        CUDA_CHECK(hipMalloc(&d_noiseEstimation, size * sizeof(double)));
        CUDA_CHECK(hipMemset(d_noiseEstimation, 0, size * sizeof(double)));
    }
    if (!d_thresholdingMap) {
        CUDA_CHECK(hipMalloc(&d_thresholdingMap, size * sizeof(double)));
        CUDA_CHECK(hipMemset(d_thresholdingMap, 0, size * sizeof(double)));
    }
    if (!d_peakList) {
        CUDA_CHECK(hipMalloc(&d_peakList, max_num_peaks * sizeof(Peak)));
        CUDA_CHECK(hipMemset(d_peakList, 0, max_num_peaks * sizeof(Peak)));
    }
}// allocate_peakInfo_mem_device
void peakInfo::free_peakInfo_device() {
    if(d_peak_counter) {
        CUDA_CHECK(hipFree(d_peak_counter));
        d_peak_counter = nullptr;
    }
    if (d_nci) {
        CUDA_CHECK(hipFree(d_nci));
        d_nci = nullptr;
    }
    if (d_foldedNci) {
        CUDA_CHECK(hipFree(d_foldedNci));
        d_foldedNci = nullptr;
    }
    if (d_noiseEstimation) {
        CUDA_CHECK(hipFree(d_noiseEstimation));
        d_noiseEstimation = nullptr;
    }
    if (d_thresholdingMap) {
        CUDA_CHECK(hipFree(d_thresholdingMap));
        d_thresholdingMap = nullptr;
    }
    if (d_peakList) {
        CUDA_CHECK(hipFree(d_peakList));
        d_peakList = nullptr;
    }
}//free_peakInfo_device
void peakInfo::copy_peakInfo_to_host() {
    int size = num_chirps * num_samples;
    if(d_peak_counter) {
        CUDA_CHECK(hipMemcpy(&num_peaks, d_peak_counter, sizeof(int), hipMemcpyDeviceToHost));
    }
    if (d_nci) {
        CUDA_CHECK(hipMemcpy(nci, d_nci, size* sizeof(double), hipMemcpyDeviceToHost));
    }
    if (d_foldedNci) {
        CUDA_CHECK(hipMemcpy(foldedNci, d_foldedNci, size * sizeof(double), hipMemcpyDeviceToHost));
    }
    if (d_noiseEstimation) {
        CUDA_CHECK(hipMemcpy(noiseEstimation, d_noiseEstimation, size * sizeof(double), hipMemcpyDeviceToHost));
    }
    if (d_thresholdingMap) {
        CUDA_CHECK(hipMemcpy(thresholdingMap, d_thresholdingMap, size * sizeof(double), hipMemcpyDeviceToHost));
    }
    if (d_peakList) {
        CUDA_CHECK(hipMemcpy(peakList, d_peakList, max_num_peaks * sizeof(Peak), hipMemcpyDeviceToHost));
    }
}
void peakInfo::initializePeakSnaps(){
    if(!peaksnaps)
    {
        peaksnaps = new Complex[num_peaks*num_receivers];
        memset(peaksnaps, 0, num_peaks * num_receivers * sizeof(Complex));
    }
    if(!d_peaksnaps) {
        CUDA_CHECK(hipMalloc(&d_peaksnaps, num_peaks * num_receivers * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemset(d_peaksnaps, 0, num_peaks * num_receivers * sizeof(hipDoubleComplex)));
    }
}
void peakInfo::freePeakSnaps() {
    if (peaksnaps) {
        delete[] peaksnaps;
        peaksnaps = nullptr;
    }
    if (d_peaksnaps) {
        CUDA_CHECK(hipFree(d_peaksnaps));
        d_peaksnaps = nullptr;
    }
} // freePeakSnaps
void peakInfo::copyPeakSnapsToHost() {
    if (d_peaksnaps) {
        CUDA_CHECK(hipMemcpy(reinterpret_cast<hipDoubleComplex*>(peaksnaps), d_peaksnaps, num_peaks * num_receivers * sizeof(Complex), hipMemcpyDeviceToHost));
    }
} // copyPeakSnapsToHost




DoAInfo::DoAInfo(int num_peaks, int num_receivers)
    : num_peaks(num_peaks), num_receivers(num_receivers), angles(nullptr), d_angles(nullptr), d_R(nullptr), 
      d_eigenvectors(nullptr), d_eigenvalues(nullptr), d_eigenvector(nullptr), d_next_eigenvector(nullptr), 
      d_noiseSubspace(nullptr), d_steeringVector(nullptr), R(nullptr), eigenvalues(nullptr), eigenvectors(nullptr){
    // Initialize any required resources
    initialize();
    
}
DoAInfo::~DoAInfo() {
    free_angles_host();
    free_angles_device();
    free_R_device();
    free_eigenData();
    free_noiseSubspace();
    free_steeringVector();
    free_R_host();
}

void DoAInfo::allocate_angles_mem_host() {
    if (!angles) {
        angles = new DoAangles[num_peaks];
        memset(angles, 0, num_peaks * sizeof(DoAangles));
    }
} // allocate_angles_mem_host

void DoAInfo::free_angles_host() {
    delete[] angles;
    angles = nullptr;
} // free_angles_host
void DoAInfo::allocate_angles_mem_device() {
    if (!d_angles) {
        CUDA_CHECK(hipMalloc(&d_angles, num_peaks * sizeof(DoAangles)));
        CUDA_CHECK(hipMemset(d_angles, 0, num_peaks * sizeof(DoAangles)));
    }
} // allocate_angles_mem_device
void DoAInfo::free_angles_device() {
    if (d_angles) {
        CUDA_CHECK(hipFree(d_angles));
        d_angles = nullptr;
    }
} // free_angles_device
void DoAInfo::copy_angles_to_host() {
    if (d_angles) {
        CUDA_CHECK(hipMemcpy(angles, d_angles, num_peaks * sizeof(DoAangles), hipMemcpyDeviceToHost));
    }
} // copy_angles_to_host
void DoAInfo::initialize() {
    allocate_angles_mem_host();
    allocate_angles_mem_device();
    allocate_R_mem_device();    
    init_eigenData();
    init_noiseSubspace();
    init_steeringVector();
    allocate_R_mem_host();
} // initialize

void DoAInfo::allocate_R_mem_device() {
    if (!d_R) {
        size_t size = num_peaks*num_receivers * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMalloc(&d_R, size));
        CUDA_CHECK(hipMemset(d_R, 0, size));
    }

} // allocate_R_mem_device
void DoAInfo::free_R_device() {
    if (d_R) {
        CUDA_CHECK(hipFree(d_R));
        d_R = nullptr;
    }
} // free_R_device

void DoAInfo::init_eigenData() {
    if (!d_eigenvectors) {
        size_t size = num_peaks*num_receivers * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMalloc(&d_eigenvectors, size));
        CUDA_CHECK(hipMemset(d_eigenvectors, 0, size));
    }
    if (!d_eigenvalues) {
        CUDA_CHECK(hipMalloc(&d_eigenvalues, num_peaks*num_receivers * sizeof(double)));
        CUDA_CHECK(hipMemset(d_eigenvalues, 0, num_peaks*num_receivers * sizeof(double)));
    }
    if (!d_eigenvector) {
        CUDA_CHECK(hipMalloc(&d_eigenvector, num_peaks*num_receivers * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemset(d_eigenvector, 0, num_peaks*num_receivers * sizeof(hipDoubleComplex)));
    }
    if (!d_next_eigenvector) {
        CUDA_CHECK(hipMalloc(&d_next_eigenvector, num_peaks*num_receivers * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemset(d_next_eigenvector, 0, num_peaks*num_receivers * sizeof(hipDoubleComplex)));
    }
    if(!eigenvalues) {
        eigenvalues = new double[num_peaks*num_receivers];
        memset(eigenvalues, 0, num_peaks*num_receivers * sizeof(double));
    }
    if(!eigenvectors) {
        eigenvectors = new double[num_peaks*num_receivers * num_receivers];
        memset(eigenvectors, 0, num_peaks*num_receivers * num_receivers * sizeof(double));
    }
} // init_eigenData
void DoAInfo::free_eigenData() {
    if (d_eigenvectors) {
        CUDA_CHECK(hipFree(d_eigenvectors));
        d_eigenvectors = nullptr;
    }
    if (d_eigenvalues) {
        CUDA_CHECK(hipFree(d_eigenvalues));
        d_eigenvalues = nullptr;
    }
    if (d_eigenvector) {
        CUDA_CHECK(hipFree(d_eigenvector));
        d_eigenvector = nullptr;
    }
    if (d_next_eigenvector) {
        CUDA_CHECK(hipFree(d_next_eigenvector));
        d_next_eigenvector = nullptr;
    }
    if (eigenvalues) {
        delete[] eigenvalues;
        eigenvalues = nullptr;
    }
} // free_eigenData

void DoAInfo::copy_eigenData_to_host() {
    if (d_eigenvectors) {
        size_t size = num_peaks*num_receivers * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMemcpy(reinterpret_cast<hipDoubleComplex*>(R), d_eigenvectors, size, hipMemcpyDeviceToHost));
    }
    if (d_eigenvalues) {
        CUDA_CHECK(hipMemcpy(eigenvalues, d_eigenvalues, num_peaks*num_receivers * sizeof(double), hipMemcpyDeviceToHost));
    }
} // copy_eigenData_to_host
void DoAInfo::init_noiseSubspace() {
    if (!d_noiseSubspace) {
        size_t size = num_peaks * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMalloc(&d_noiseSubspace, size));
        CUDA_CHECK(hipMemset(d_noiseSubspace, 0, size));
    }
} // init_noiseSubspace
void DoAInfo::free_noiseSubspace() {
    if (d_noiseSubspace) {
        CUDA_CHECK(hipFree(d_noiseSubspace));
        d_noiseSubspace = nullptr;
    }
} // free_noiseSubspace
void DoAInfo::init_steeringVector() {
    if (!d_steeringVector) {
        size_t size = num_peaks * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMalloc(&d_steeringVector, size));
        CUDA_CHECK(hipMemset(d_steeringVector, 0, size));
    }
} // init_steeringVector
void DoAInfo::free_steeringVector() {
    if (d_steeringVector) {
        CUDA_CHECK(hipFree(d_steeringVector));
        d_steeringVector = nullptr;
    }
} // free_steeringVector
void DoAInfo::copy_R_to_host() {
    if (d_R) {
        size_t size = num_peaks*num_receivers * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMemcpy(reinterpret_cast<hipDoubleComplex*>(R), d_R, size, hipMemcpyDeviceToHost));
    }
} // copy_R_to_host
void DoAInfo::allocate_R_mem_host() {
    if (!R) {
        R = new Complex[num_peaks*num_receivers * num_receivers];
        memset(R, 0, num_peaks*num_receivers * num_receivers * sizeof(Complex));
    }
} // allocate_R_mem_host
void DoAInfo::free_R_host() {
    delete[] R;
    R = nullptr;
} // free_R_host

TargetResults::TargetResults(int max_targets)
    : targets(nullptr), d_targets(nullptr), num_targets(max_targets) {
    allocate_host(max_targets);
    allocate_device(max_targets);
}

TargetResults::~TargetResults() {
    free_host();
    free_device();
}

void TargetResults::allocate_host(int max_targets) {
    if (!targets) {
        targets = new Target[max_targets];
        memset(targets, 0, max_targets * sizeof(Target));
        for (int i = 0; i < max_targets; ++i) {
            targets[i].rcs = 0.0;
        }
    }
}

void TargetResults::allocate_device(int max_targets) {
    if (!d_targets) {
        CUDA_CHECK(hipMalloc(&d_targets, max_targets * sizeof(Target)));
        CUDA_CHECK(hipMemset(d_targets, 0, max_targets * sizeof(Target)));
        // Optionally set rcs to 0.0 in device memory (not strictly needed after memset)
    }
}

void TargetResults::free_host() {
    if (targets) {
        delete[] targets;
        targets = nullptr;
    }
}

void TargetResults::free_device() {
    if (d_targets) {
        CUDA_CHECK(hipFree(d_targets));
        d_targets = nullptr;
    }
}

void TargetResults::copy_to_host() {
    if (d_targets && targets) {
        CUDA_CHECK(hipMemcpy(targets, d_targets, num_targets * sizeof(Target), hipMemcpyDeviceToHost));
    }
}

/**
 * Free all GPU memory resources associated with radar data structures
 * 
 * This function centralizes all memory cleanup operations for radar data structures
 * to ensure consistent and complete memory management.
 * 
 * @param frame Pointer to radar frame structure to cleanup (nullptr to skip)
 * @param peakinfo Pointer to peak detection information structure to cleanup (nullptr to skip)
 * @param doaInfo Pointer to direction of arrival information structure to cleanup (nullptr to skip)
 * @param targetResults Pointer to target processing results structure to cleanup (nullptr to skip)
 * @param cleanupFrame Whether to clean up frame resources (default: true)
 * @param cleanupPeakInfo Whether to clean up peak info resources (default: true)
 */
void cleanupRadarResources(
    Frame* frame,
    peakInfo* peakinfo,
    DoAInfo* doaInfo,
    TargetResults* targetResults,
    bool cleanupFrame,
    bool cleanupPeakInfo
) {
    // Clean up frame resources if provided and flag is set
    if (frame && cleanupFrame) {
        frame->free_device();
    }
    
    // Clean up peak detection resources if provided and flag is set
    if (peakinfo && cleanupPeakInfo) {
        peakinfo->free_peakInfo_device();
    }
    
    // Clean up DoA processing resources if provided
    if (doaInfo) {
        doaInfo->free_angles_device();
        doaInfo->free_R_device();
        doaInfo->free_eigenData();
        doaInfo->free_noiseSubspace();
        doaInfo->free_steeringVector();
    }
    
    // Clean up target processing resources if provided
    if (targetResults) {
        targetResults->free_device();
    }
    
    // Output memory cleanup confirmation
    std::cout << "Radar processing resources cleaned up successfully" << std::endl;
}

// Overload for reference parameters (for backward compatibility)
void cleanupRadarResources(
    Frame& frame,
    peakInfo& peakinfo,
    DoAInfo& doaInfo,
    TargetResults& targetResults
) {
    cleanupRadarResources(&frame, &peakinfo, &doaInfo, &targetResults, true, true);
}

/**
 * Clean up all resources associated with batch processing
 * 
 * This function centralizes all memory cleanup operations for batch processing,
 * including persistent arrays, frame arrays, and other radar data structures.
 * 
 * @param frames Vector of radar frames to clean up
 * @param peakInfos Vector of peak detection information structures to clean up
 * @param doaInfo Direction of arrival information structure to clean up
 * @param targetResults Target processing results structure to clean up
 * @param persistentArraysInitialized Whether persistent arrays have been initialized
 * @param cleanupPersistentArrays Function pointer to cleanup persistent arrays
 */
void cleanupBatchResources(
    std::vector<Frame>& frames,
    std::vector<peakInfo>& peakInfos,
    DoAInfo& doaInfo,
    TargetResults& targetResults,
    bool persistentArraysInitialized,
    void (*cleanupPersistentArrays)()
) {
    // First, clean up any persistent arrays used for batch processing
    if (persistentArraysInitialized && cleanupPersistentArrays) {
        std::cout << "Cleaning up persistent arrays for batch peak detection..." << std::endl;
        cleanupPersistentArrays();
    }
    
    // Clean up frame resources
    for (auto& frame : frames) {
        frame.free_device();
    }
    
    // Clean up peak info resources
    for (auto& pi : peakInfos) {
        pi.free_peakInfo_device();
        pi.free_peakInfo_host();
    }
    
    // Clean up other shared resources
    doaInfo.free_angles_device();
    doaInfo.free_R_device();
    doaInfo.free_eigenData();
    doaInfo.free_noiseSubspace();
    doaInfo.free_steeringVector();
    
    // Clean up target results
    targetResults.free_device();
    
    std::cout << "Batch processing resources cleaned up successfully" << std::endl;
}

} // namespace RadarData