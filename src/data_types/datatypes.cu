#include "datatypes.cuh"
#include <fstream>
#include <sstream>
#include <iostream>
#include <cstring>
#include "../cuda_utils/cuda_utils.hpp"

namespace RadarData {

Frame::Frame(int r, int c, int s)
    : num_receivers(r), num_chirps(c), num_samples(s), d_data(nullptr)
{
    data = new Complex[r * c * s]();
    allocate_frame_mem_device();
}

Frame::~Frame() {
    delete[] data;
    free_device();
}

Complex& Frame::operator()(int receiver, int chirp, int sample) {
    return data[idx(receiver, chirp, sample)];
}
const Complex& Frame::operator()(int receiver, int chirp, int sample) const {
    return data[idx(receiver, chirp, sample)];
}

// Device memory management
void Frame::allocate_frame_mem_device() {
    if (!d_data) {
        size_t total = num_receivers * num_chirps * num_samples;
        CUDA_CHECK(hipMalloc(&d_data, total * sizeof(hipDoubleComplex)));
    }
}
void Frame::free_device() {
    if (d_data) {
        CUDA_CHECK(hipFree(d_data));
        d_data = nullptr;
    }
}
void Frame::copy_frame_to_device() {
    size_t total = num_receivers * num_chirps * num_samples;
    CUDA_CHECK(hipMemcpy(
    d_data,
    reinterpret_cast<const hipDoubleComplex*>(data),
    total * sizeof(hipDoubleComplex),
    hipMemcpyHostToDevice));
    //std::cout << "Frame Data copied to device" << std::endl;
}
void Frame::copy_frame_to_host() {
    if (d_data) {
        size_t total = num_receivers * num_chirps * num_samples;
       CUDA_CHECK(hipMemcpy(
    reinterpret_cast<hipDoubleComplex*>(data),
    d_data,
    total * sizeof(hipDoubleComplex),
    hipMemcpyDeviceToHost));
    }
}

// Initialize frame with data from CSV
void initialize_frame(Frame& frame, int num_receivers, int num_chirps, int num_samples, int frameIndex) {
    //Frame frame(num_receivers, num_chirps, num_samples);

    std::ifstream file("/mnt/mydisk/Nilesh/CUDA_RSP/data/radar_indexed.csv");
    if (!file.is_open()) {
        std::cerr << "Error: Could not open radar_indexed.csv" << std::endl;
        return;
    }

    std::string line;
    bool frameDataLoaded = false;
    while (std::getline(file, line)) {
        std::istringstream ss(line);
        int frame_number, receiver, chirp, sample;
        double value;
        char delimiter;
        ss >> frame_number >> delimiter >> receiver >> delimiter >> chirp >> delimiter >> sample >> delimiter >> value;

        if (frame_number == frameIndex) {
            if (receiver < num_receivers && chirp < num_chirps && sample < num_samples) {
                frame(receiver, chirp, sample) = Complex(value, 0);
            }
            frameDataLoaded = true;
        } else if (frameDataLoaded) {
            break;
        }
    }
    file.close();
    //return frame;
}

size_t frame_size_bytes(const Frame& frame) {
    return static_cast<size_t>(frame.num_receivers) *
           frame.num_chirps *
           frame.num_samples *
           sizeof(Complex);
}
peakInfo::peakInfo(int r, int c, int s)
{
    num_receivers = r;
    num_chirps = c;
    num_samples = s;
    std::cout << "Creating peakInfo with dimensions: "
              << num_receivers << " receivers, "
              << num_chirps << " chirps, "
              << num_samples << " samples." << std::endl;
    value = 0.0;
    num_peaks = 0; // Initialize number of peaks to zero
     // Device variable to hold number of peaks
    max_num_peaks = num_receivers*num_chirps*num_samples; // Default value, can be adjusted as needed
    
    nci = nullptr;
    foldedNci = nullptr;
    noiseEstimation = nullptr;
    thresholdingMap = nullptr;
    peakList = nullptr;
    peaksnaps = nullptr;    
    
    d_nci = nullptr;
    d_foldedNci = nullptr;
    d_noiseEstimation = nullptr;
    d_thresholdingMap = nullptr;
    d_peakList = nullptr;
    d_num_peaks = nullptr;
    d_peak_counter = nullptr;
    d_peaksnaps = nullptr;
    allocate_peakInfo_mem_host();
    allocate_peakInfo_mem_device();
}
peakInfo::~peakInfo() {
    free_peakInfo_host();
    free_peakInfo_device();
}
void peakInfo::allocate_peakInfo_mem_host() {
    int size = num_chirps * num_samples;
    //std::cout << "Allocating memory for peakInfo on host: " << size << " elements." << std::endl;
    if (!nci) {
        nci = new double[size];
        memset(nci, 0, size * sizeof(double));
    }
    if (!foldedNci) {
        foldedNci = new double[size];
        memset(foldedNci, 0, size * sizeof(double));
    }
    if (!noiseEstimation) {
        noiseEstimation = new double[size];
        memset(noiseEstimation, 0, size * sizeof(double));
    }
    if (!thresholdingMap) {
        thresholdingMap = new double[size];
        memset(thresholdingMap, 0, size * sizeof(double));
    }
    if (!peakList) {
        peakList = new Peak[max_num_peaks];
        memset(peakList, 0, max_num_peaks * sizeof(Peak));
    }
} //allocate_peakInfo_mem_host
void peakInfo::free_peakInfo_host() {
    delete[] nci;
    delete[] foldedNci;
    delete[] noiseEstimation;
    delete[] thresholdingMap;
    delete[] peakList;

    nci = nullptr;
    foldedNci = nullptr;
    noiseEstimation = nullptr;
    thresholdingMap = nullptr;
    peakList = nullptr;
}// free_peakInfo_host

void peakInfo::allocate_peakInfo_mem_device() {
    int size = num_chirps * num_samples;
    if(!d_peak_counter){
        CUDA_CHECK(hipMalloc(&d_peak_counter, sizeof(int)));
        CUDA_CHECK(hipMemset(d_peak_counter, 0, sizeof(int)));
    }
    if (!d_nci) {
        CUDA_CHECK(hipMalloc(&d_nci, size * sizeof(double)));
        CUDA_CHECK(hipMemset(d_nci, 0, size * sizeof(double)));
    }
    if (!d_foldedNci) {
        CUDA_CHECK(hipMalloc(&d_foldedNci, size * sizeof(double)));
        CUDA_CHECK(hipMemset(d_foldedNci, 0, size * sizeof(double)));
    }
    if (!d_noiseEstimation) {
        CUDA_CHECK(hipMalloc(&d_noiseEstimation, size * sizeof(double)));
        CUDA_CHECK(hipMemset(d_noiseEstimation, 0, size * sizeof(double)));
    }
    if (!d_thresholdingMap) {
        CUDA_CHECK(hipMalloc(&d_thresholdingMap, size * sizeof(double)));
        CUDA_CHECK(hipMemset(d_thresholdingMap, 0, size * sizeof(double)));
    }
    if (!d_peakList) {
        CUDA_CHECK(hipMalloc(&d_peakList, max_num_peaks * sizeof(Peak)));
        CUDA_CHECK(hipMemset(d_peakList, 0, max_num_peaks * sizeof(Peak)));
    }
}// allocate_peakInfo_mem_device
void peakInfo::free_peakInfo_device() {
    if(d_peak_counter) {
        CUDA_CHECK(hipFree(d_peak_counter));
        d_peak_counter = nullptr;
    }
    if (d_nci) {
        CUDA_CHECK(hipFree(d_nci));
        d_nci = nullptr;
    }
    if (d_foldedNci) {
        CUDA_CHECK(hipFree(d_foldedNci));
        d_foldedNci = nullptr;
    }
    if (d_noiseEstimation) {
        CUDA_CHECK(hipFree(d_noiseEstimation));
        d_noiseEstimation = nullptr;
    }
    if (d_thresholdingMap) {
        CUDA_CHECK(hipFree(d_thresholdingMap));
        d_thresholdingMap = nullptr;
    }
    if (d_peakList) {
        CUDA_CHECK(hipFree(d_peakList));
        d_peakList = nullptr;
    }
}//free_peakInfo_device
void peakInfo::copy_peakInfo_to_host() {
    int size = num_chirps * num_samples;
    if(d_peak_counter) {
        CUDA_CHECK(hipMemcpy(&num_peaks, d_peak_counter, sizeof(int), hipMemcpyDeviceToHost));
    }
    if (d_nci) {
        CUDA_CHECK(hipMemcpy(nci, d_nci, size* sizeof(double), hipMemcpyDeviceToHost));
    }
    if (d_foldedNci) {
        CUDA_CHECK(hipMemcpy(foldedNci, d_foldedNci, size * sizeof(double), hipMemcpyDeviceToHost));
    }
    if (d_noiseEstimation) {
        CUDA_CHECK(hipMemcpy(noiseEstimation, d_noiseEstimation, size * sizeof(double), hipMemcpyDeviceToHost));
    }
    if (d_thresholdingMap) {
        CUDA_CHECK(hipMemcpy(thresholdingMap, d_thresholdingMap, size * sizeof(double), hipMemcpyDeviceToHost));
    }
    if (d_peakList) {
        CUDA_CHECK(hipMemcpy(peakList, d_peakList, max_num_peaks * sizeof(Peak), hipMemcpyDeviceToHost));
    }
}
void peakInfo::initializePeakSnaps(){
    if(!peaksnaps)
    {
        peaksnaps = new Complex[num_peaks*num_receivers];
        memset(peaksnaps, 0, num_peaks * num_receivers * sizeof(Complex));
    }
    if(!d_peaksnaps) {
        CUDA_CHECK(hipMalloc(&d_peaksnaps, num_peaks * num_receivers * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemset(d_peaksnaps, 0, num_peaks * num_receivers * sizeof(hipDoubleComplex)));
    }
}
void peakInfo::freePeakSnaps() {
    if (peaksnaps) {
        delete[] peaksnaps;
        peaksnaps = nullptr;
    }
    if (d_peaksnaps) {
        CUDA_CHECK(hipFree(d_peaksnaps));
        d_peaksnaps = nullptr;
    }
} // freePeakSnaps
void peakInfo::copyPeakSnapsToHost() {
    if (d_peaksnaps) {
        CUDA_CHECK(hipMemcpy(reinterpret_cast<hipDoubleComplex*>(peaksnaps), d_peaksnaps, num_peaks * num_receivers * sizeof(Complex), hipMemcpyDeviceToHost));
    }
} // copyPeakSnapsToHost




DoAInfo::DoAInfo(int num_peaks, int num_receivers)
    : num_peaks(num_peaks), num_receivers(num_receivers), angles(nullptr), d_angles(nullptr), d_R(nullptr), 
      d_eigenvectors(nullptr), d_eigenvalues(nullptr), d_eigenvector(nullptr), d_next_eigenvector(nullptr), 
      d_noiseSubspace(nullptr), d_steeringVector(nullptr), R(nullptr), eigenvalues(nullptr), eigenvectors(nullptr){
    // Initialize any required resources
    initialize();
    
}
DoAInfo::~DoAInfo() {
    free_angles_host();
    free_angles_device();
    free_R_device();
    free_eigenData();
    free_noiseSubspace();
    free_steeringVector();
    free_R_host();
}

void DoAInfo::allocate_angles_mem_host() {
    if (!angles) {
        angles = new DoAangles[num_peaks];
        memset(angles, 0, num_peaks * sizeof(DoAangles));
    }
} // allocate_angles_mem_host

void DoAInfo::free_angles_host() {
    delete[] angles;
    angles = nullptr;
} // free_angles_host
void DoAInfo::allocate_angles_mem_device() {
    if (!d_angles) {
        CUDA_CHECK(hipMalloc(&d_angles, num_peaks * sizeof(DoAangles)));
        CUDA_CHECK(hipMemset(d_angles, 0, num_peaks * sizeof(DoAangles)));
    }
} // allocate_angles_mem_device
void DoAInfo::free_angles_device() {
    if (d_angles) {
        CUDA_CHECK(hipFree(d_angles));
        d_angles = nullptr;
    }
} // free_angles_device
void DoAInfo::copy_angles_to_host() {
    if (d_angles) {
        CUDA_CHECK(hipMemcpy(angles, d_angles, num_peaks * sizeof(DoAangles), hipMemcpyDeviceToHost));
    }
} // copy_angles_to_host
void DoAInfo::initialize() {
    allocate_angles_mem_host();
    allocate_angles_mem_device();
    allocate_R_mem_device();    
    init_eigenData();
    init_noiseSubspace();
    init_steeringVector();
    allocate_R_mem_host();
} // initialize

void DoAInfo::allocate_R_mem_device() {
    if (!d_R) {
        size_t size = num_peaks*num_receivers * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMalloc(&d_R, size));
        CUDA_CHECK(hipMemset(d_R, 0, size));
    }

} // allocate_R_mem_device
void DoAInfo::free_R_device() {
    if (d_R) {
        CUDA_CHECK(hipFree(d_R));
        d_R = nullptr;
    }
} // free_R_device

void DoAInfo::init_eigenData() {
    if (!d_eigenvectors) {
        size_t size = num_peaks*num_receivers * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMalloc(&d_eigenvectors, size));
        CUDA_CHECK(hipMemset(d_eigenvectors, 0, size));
    }
    if (!d_eigenvalues) {
        CUDA_CHECK(hipMalloc(&d_eigenvalues, num_peaks*num_receivers * sizeof(double)));
        CUDA_CHECK(hipMemset(d_eigenvalues, 0, num_peaks*num_receivers * sizeof(double)));
    }
    if (!d_eigenvector) {
        CUDA_CHECK(hipMalloc(&d_eigenvector, num_peaks*num_receivers * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemset(d_eigenvector, 0, num_peaks*num_receivers * sizeof(hipDoubleComplex)));
    }
    if (!d_next_eigenvector) {
        CUDA_CHECK(hipMalloc(&d_next_eigenvector, num_peaks*num_receivers * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMemset(d_next_eigenvector, 0, num_peaks*num_receivers * sizeof(hipDoubleComplex)));
    }
    if(!eigenvalues) {
        eigenvalues = new double[num_peaks*num_receivers];
        memset(eigenvalues, 0, num_peaks*num_receivers * sizeof(double));
    }
    if(!eigenvectors) {
        eigenvectors = new double[num_peaks*num_receivers * num_receivers];
        memset(eigenvectors, 0, num_peaks*num_receivers * num_receivers * sizeof(double));
    }
} // init_eigenData
void DoAInfo::free_eigenData() {
    if (d_eigenvectors) {
        CUDA_CHECK(hipFree(d_eigenvectors));
        d_eigenvectors = nullptr;
    }
    if (d_eigenvalues) {
        CUDA_CHECK(hipFree(d_eigenvalues));
        d_eigenvalues = nullptr;
    }
    if (d_eigenvector) {
        CUDA_CHECK(hipFree(d_eigenvector));
        d_eigenvector = nullptr;
    }
    if (d_next_eigenvector) {
        CUDA_CHECK(hipFree(d_next_eigenvector));
        d_next_eigenvector = nullptr;
    }
    if (eigenvalues) {
        delete[] eigenvalues;
        eigenvalues = nullptr;
    }
} // free_eigenData

void DoAInfo::copy_eigenData_to_host() {
    if (d_eigenvectors) {
        size_t size = num_peaks*num_receivers * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMemcpy(reinterpret_cast<hipDoubleComplex*>(R), d_eigenvectors, size, hipMemcpyDeviceToHost));
    }
    if (d_eigenvalues) {
        CUDA_CHECK(hipMemcpy(eigenvalues, d_eigenvalues, num_peaks*num_receivers * sizeof(double), hipMemcpyDeviceToHost));
    }
} // copy_eigenData_to_host
void DoAInfo::init_noiseSubspace() {
    if (!d_noiseSubspace) {
        size_t size = num_peaks * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMalloc(&d_noiseSubspace, size));
        CUDA_CHECK(hipMemset(d_noiseSubspace, 0, size));
    }
} // init_noiseSubspace
void DoAInfo::free_noiseSubspace() {
    if (d_noiseSubspace) {
        CUDA_CHECK(hipFree(d_noiseSubspace));
        d_noiseSubspace = nullptr;
    }
} // free_noiseSubspace
void DoAInfo::init_steeringVector() {
    if (!d_steeringVector) {
        size_t size = num_peaks * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMalloc(&d_steeringVector, size));
        CUDA_CHECK(hipMemset(d_steeringVector, 0, size));
    }
} // init_steeringVector
void DoAInfo::free_steeringVector() {
    if (d_steeringVector) {
        CUDA_CHECK(hipFree(d_steeringVector));
        d_steeringVector = nullptr;
    }
} // free_steeringVector
void DoAInfo::copy_R_to_host() {
    if (d_R) {
        size_t size = num_peaks*num_receivers * num_receivers * sizeof(hipDoubleComplex);
        CUDA_CHECK(hipMemcpy(reinterpret_cast<hipDoubleComplex*>(R), d_R, size, hipMemcpyDeviceToHost));
    }
} // copy_R_to_host
void DoAInfo::allocate_R_mem_host() {
    if (!R) {
        R = new Complex[num_peaks*num_receivers * num_receivers];
        memset(R, 0, num_peaks*num_receivers * num_receivers * sizeof(Complex));
    }
} // allocate_R_mem_host
void DoAInfo::free_R_host() {
    delete[] R;
    R = nullptr;
} // free_R_host

TargetResults::TargetResults(int max_targets)
    : targets(nullptr), d_targets(nullptr), num_targets(max_targets) {
    allocate_host(max_targets);
    allocate_device(max_targets);
}

TargetResults::~TargetResults() {
    free_host();
    free_device();
}

void TargetResults::allocate_host(int max_targets) {
    if (!targets) {
        targets = new Target[max_targets];
        memset(targets, 0, max_targets * sizeof(Target));
    }
}

void TargetResults::allocate_device(int max_targets) {
    if (!d_targets) {
        CUDA_CHECK(hipMalloc(&d_targets, max_targets * sizeof(Target)));
        CUDA_CHECK(hipMemset(d_targets, 0, max_targets * sizeof(Target)));
    }
}

void TargetResults::free_host() {
    if (targets) {
        delete[] targets;
        targets = nullptr;
    }
}

void TargetResults::free_device() {
    if (d_targets) {
        CUDA_CHECK(hipFree(d_targets));
        d_targets = nullptr;
    }
}

void TargetResults::copy_to_host() {
    if (d_targets && targets) {
        CUDA_CHECK(hipMemcpy(targets, d_targets, num_targets * sizeof(Target), hipMemcpyDeviceToHost));
    }
}

} // namespace RadarData